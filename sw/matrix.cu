
#include <hip/hip_runtime.h>
//Matrix multiplication kernel with non-square matrix
__global__ void gpu_mult(double *a, double *b, double *c, int M, int N, int K)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if(row < M && col < K)
  {
    int temp = 0;
    for(int i = 0; i < N; i++)
    {
        temp += a[row * N + i] * b[i * K + col];
    }
    c[row * K + col] = temp;
  }
}