#include "hip/hip_runtime.h"
#include "stdlib.h"
#include "sequence_gpu.h"
#include "hip/hip_runtime.h"
#include ""
#include <cassert>
#include <cstddef>
#include <cstdlib>
#include <iostream>
#include <ostream>
#include <unordered_map>
#include <hip/hip_runtime.h>


__global__ void matrix_mult_gpu(int *a, int *b, int *c, int M, int N, int K)
{
    // Calculate row and column
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < M && col < K)
    {
        // Temp variable for acuumulatting results
        int temp = 0;
        for(int i = 0; i < N; i++)
        {
            temp += a[row * N + i] * b[i * K + col];
        }
        c[row * K + col] = temp;
    }
}


Sequence::Sequence(
    std::vector<std::vector<int>> temp_table,
    std::unordered_map<char,matrix*> &temp_dict) : s_table(temp_table), str_matrix_dict(temp_dict)
{
}   


Node* Sequence::init_sequence(int i, int j)
{
    /*
        Base cases
    */
    if(i > j) 
    {
        return nullptr;
    }
    Node* n = new Node();
    if(i == j-1)
    {
        n->left = nullptr;
        n->right = nullptr;
        n->seq = static_cast<char*>(malloc(3 * sizeof(char)));
        n->seq[0] = '0' + i;
        n->seq[1] = '0' + j;
        n->seq[2] = '\0';
        return n;
    }
    if(i == j)
    {
        //Node n;
        n->left = nullptr;
        n->right = nullptr;
        n->seq = static_cast<char*>(malloc(2 * sizeof(char)));
        n->seq[0] = '0' + i;
        n->seq[1] = '\0';
        return n;
    }
    int sub_sequence = s_table[i-1][j-1];
    n->seq = static_cast<char*>(malloc((j-i+1) * sizeof(char)));
    for(int x = 0; x <= (j-i); x++)
    {
        n->seq[x] = '0' + (x + i);
    }
    n->seq[j-i+1] = '\0';
    n->left = init_sequence(i, sub_sequence);
    n->right = init_sequence(sub_sequence+1, j);
    return n;
}

void Sequence::print_sequence(Node* n)
{
    if(n == nullptr)
    {
        return;
    }
    if(n->left == nullptr && n->right == nullptr)
    {
        std::cout << n->seq << " ";
    }
    
    if(n->left)
    {
        std::cout << "(" << " ";
        print_sequence(n->left);
        std::cout << ")" << " ";
    }
    if(n->right)
    {
        std::cout << "(" << " ";
        print_sequence(n->right);
        std::cout << ")" << " ";
    }
}

void Sequence::printMatrix(matrix *m)
{
    int rows, cols;
    std::tie(rows, cols) = m->dimension;
    for(int i = 0; i < rows; i++)
    {
        for(int j = 0; j < cols; j++)
        {
            std::cout << m->values[i][j] << " ";
        }
        std::cout << std::endl;
    }
}


matrix* Sequence::compute(Node* n)
{
    /*
        Solo
    */
    if(n->left == nullptr && n->right == nullptr && n->seq[1] == '\0') 
    {
        return str_matrix_dict[n->seq[0]];
    }
    /*
        Two pairs
    */
    if(n->left == nullptr && n->right == nullptr && n->seq[2] == '\0')
    {
        matrix* matrix_A = str_matrix_dict[n->seq[0]];
        matrix* matrix_B = str_matrix_dict[n->seq[1]];
        matrix* matrix_C = new matrix;
        int m,n,x,y;
        std::tie(m,n) = matrix_A->dimension;
        std::tie(x,y) = matrix_B->dimension;
        matrix_C->dimension = std::make_tuple(m,y);
        matrix_C->values.resize(m);
        for (int i = 0; i < m; ++i) 
        {
            matrix_C->values[i].resize(y);
        }
        int val = matrix_A->values[0][0];
        matrix_mult(matrix_A,matrix_B,matrix_C,m,y,x);
        return matrix_C;
    }
    else
    {
        matrix* left_res = compute(n->left);
        matrix* right_res = compute(n->right);

        matrix* matrix_C = new matrix;
        int a,b,c,d;
        std::tie(a,b) = left_res->dimension;
        std::tie(c,d) = right_res->dimension;
        matrix_C->dimension = std::make_tuple(a,d);
        matrix_C->values.resize(a);
        for (int i = 0; i < a; ++i) 
        {
            matrix_C->values[i].resize(d);
        }
        matrix_mult(left_res,right_res,matrix_C,a,d,b);
        return matrix_C;   
    }  
}



void Sequence::matrix_mult(matrix *a, matrix *b, matrix *c, int x, int y, int z)
{
    for(int row = 0; row < x; row++)
    {
        for(int col = 0; col < y; col++)
        {
            for(int k = 0; k < z; k++)
            {
                c->values[row][col] += (a->values[row][k] * b->values[k][col]);
            }
        }
    }
}




void Sequence::init_dp(std::vector<matrix> list_matrixes)
{
    /*
        Create dictionary for M_table called 'seen'

        -In C++ you need to provide a hashing function if you are 
        using a custom key 
    */
    std::unordered_map<std::pair<int,int>,int, pair_hash> seen;
    for(int i = 0; i < list_matrixes.size(); i++)
    {
        seen[std::make_pair(i,i)] = 0;
    }

    /*
        Extract D_0 -> D_(N-1) dimenison values
    */
    std::vector<int> dimenions;
    for(int i = 0; i < list_matrixes.size(); i++)
    {
        matrix temp = list_matrixes[i];
        int a = std::get<0>(temp.dimension);
        int b = std::get<1>(temp.dimension);
        if(i == list_matrixes.size()-1)
        {
            dimenions.push_back(a);
            dimenions.push_back(b); 
        }
        else
        {
            dimenions.push_back(a);
        }
    }
     dp(1,list_matrixes.size(), dimenions, &seen);
    //std::cout << "MAX is " << seen[{1,list_matrixes.size()}] << std::endl;
}

int Sequence::dp(int i, int j, std::vector<int> dimensions, std::unordered_map<std::pair<int, int>, int, pair_hash> *seen)
{
     /*
        Base cases
    */
    if(i == j)
    {
        return 0;
    }
    std::pair<int,int> pair = std::make_pair(i,j);
    if(seen->find((pair)) != seen->end())
    {
        return (*seen)[pair];    
    }
    int res = 1000000;
    int s_value = 0;
    for(int k = i; k < j; k++)
    {
        int x = dp(i,k,dimensions, seen);
        int y = dp(k+1, j,dimensions, seen);
        int z = dimensions[i-1] * dimensions[k] * dimensions[j];
        int ans = x + y + z;
        if(ans < res)
        {
            res = ans;
            s_value = k;
        }
    }
    s_table[i-1][j-1] = s_value;
    (*seen)[pair] = res;
    return res;
}

void Sequence::print_s_table()
{
    std::cout << "S TABLE: " << std::endl;
    for(int i = 0; i < s_table.size(); i++)
    {
        for(int j = 0; j < s_table[0].size(); j++)
        {
            std::cout << s_table[i][j] << "\t";
        }
        std::cout << std::endl;
    }
}

matrix* Sequence::gpu_compute(Node *n)
{
    /*
        Solo
    */
    if(n->left == nullptr && n->right == nullptr && n->seq[1] == '\0') 
    {
        return str_matrix_dict[n->seq[0]];
    }
    /*
        Two pairs
    */
    if(n->left == nullptr && n->right == nullptr && n->seq[2] == '\0')
    {
        matrix* matrix_A = str_matrix_dict[n->seq[0]];
        matrix* matrix_B = str_matrix_dict[n->seq[1]];
        matrix* matrix_C = new matrix;

        std::cout << "Matrix " << matrix_A->name << std::endl;
        printMatrix(matrix_A);
        std::cout << "Matrix " << matrix_B->name << std::endl;
        printMatrix(matrix_B);
        matrix_C->name = matrix_A->name + matrix_B->name;
        // Determine size of dimenisons
        int M;
        int N_1;
        int N_2;
        int K;

        std::tie(M,N_1) = matrix_A->dimension;
        std::tie(N_2,K) = matrix_B->dimension;

        assert(N_1 == N_2);
        matrix_C->dimension = std::make_tuple(M,K);
        matrix_C->values.resize(M);
        for (int i = 0; i < M; ++i) 
        {
            matrix_C->values[i].resize(K);
        }


        // Create and allocate host ptr
        int *host_memory_a = new int [M * N_1];
        int *host_memory_b = new int [N_2 * K];
        int *host_memory_c = new int [M * K];

        //transfer to int ptr
        transfer_to_ptr(host_memory_a, matrix_A);
        transfer_to_ptr(host_memory_b, matrix_B);

        std::cout << "after transfer to ptr A " << std::endl;
        int counter = 0;
        while(host_memory_a[counter] != 0)
        {
            std::cout << host_memory_a[counter] << " ";
            counter++;
        }
        std::cout << std::endl;
        std::cout << "---" << std::endl;
        std::cout << "after transfer to ptr B " << std::endl;
        counter = 0;
        while(host_memory_a[counter] != 0)
        {
            std::cout << host_memory_b[counter] << " ";
            counter++;
        }


        // Create and allocate device ptr
        int *device_memory_a, *device_memory_b, *device_memory_c;
        hipMalloc(&device_memory_a,M * N_1 * sizeof(int));
        hipMalloc(&device_memory_b,N_2 * K * sizeof(int));
        hipMalloc(&device_memory_c,M * K * sizeof(int));

        // # of threads 
        int THREADS_M = 16;
        
        // set up blocks
        int block_rows = (M + THREADS_M - 1) / THREADS_M;
        int block_cols = (K + THREADS_M - 1) / THREADS_M;

        dim3 threads_m(THREADS_M, THREADS_M);
        dim3 grid_m(block_cols, block_rows);

        // Copy data to device
        hipMemcpy(device_memory_a, host_memory_a, M * N_1 * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(device_memory_b, host_memory_b, N_2 * K * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(device_memory_c, host_memory_c, M * K * sizeof(int), hipMemcpyHostToDevice);
        
        // Call Kernel Function
        matrix_mult_gpu<<<grid_m, threads_m>>>(device_memory_a, device_memory_b, device_memory_c, M, N_1, K);

        // Copy back the results
        hipMemcpy(host_memory_c, device_memory_c, M * K * sizeof(int), hipMemcpyDeviceToHost);

        // Verify 
        transfer_to_matrix(host_memory_c, matrix_C);
        std::cout << "Matrix " << matrix_C->name << std::endl;
        printMatrix(matrix_C);
        
        free(host_memory_a);
        free(host_memory_b);
        free(host_memory_c);
        hipFree(device_memory_a);
        hipFree(device_memory_b);
        hipFree(device_memory_c);


        return matrix_C;
    }
    else
    {
        matrix* left_res = gpu_compute(n->left);
        matrix* right_res = gpu_compute(n->right);
        matrix* matrix_C = new matrix;
        
        std::cout << "Matrix left_res " << left_res->name << std::endl;
        printMatrix(left_res);
        std::cout << "Matrix right_res " << right_res->name << std::endl;
        printMatrix(right_res);
        matrix_C->name = left_res->name + right_res->name;
        // Determine size of dimenisons
        int M;
        int N_1;
        int N_2;
        int K;

        std::tie(M,N_1) = left_res->dimension;
        std::tie(N_2,K) = right_res->dimension;
        assert(N_1 == N_2);
        matrix_C->dimension = std::make_tuple(M,K);
        matrix_C->values.resize(M);
        for (int i = 0; i < M; ++i) 
        {
            matrix_C->values[i].resize(K);
        }

        // Create and allocate host ptr
        int *host_memory_a = new int [M * N_1];
        int *host_memory_b = new int [N_2 * K];
        int *host_memory_c = new int [M * K];

        // Transfer to ptr
        transfer_to_ptr(host_memory_a, left_res);
        transfer_to_ptr(host_memory_b, right_res);
        std::cout << "after transfer to ptr A " << std::endl;
        int counter = 0;
        while(host_memory_a[counter] != 0)
        {
            std::cout << host_memory_a[counter] << " ";
            counter++;
        }
        std::cout << std::endl;
        std::cout << "----" << std::endl;
        std::cout << "after transfer to ptr B " << std::endl;
        counter = 0;
        while(host_memory_a[counter] != 0)
        {
            std::cout << host_memory_b[counter] << " ";
            counter++;
        }
        // Create and allocate device ptr
        int *device_memory_a, *device_memory_b, *device_memory_c;
        hipMalloc(&device_memory_a,M * N_1 * sizeof(int));
        hipMalloc(&device_memory_b,N_2 * K * sizeof(int));
        hipMalloc(&device_memory_c,M * K * sizeof(int));

        // # of threads 
        int THREADS_M = 16;
        
        // set up blocks
        int block_row = (M + THREADS_M - 1) / THREADS_M;
        int block_cols = (K + THREADS_M - 1) / THREADS_M;

        dim3 threads_m(THREADS_M, THREADS_M);
        dim3 grid_m(block_cols, block_row);

        // Copy data to device
        hipMemcpy(device_memory_a, host_memory_a, M * N_1 * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(device_memory_b, host_memory_b, N_2 * K * sizeof(int), hipMemcpyHostToDevice);
        //hipMemcpy(device_memory_c, host_memory_c, M * K * sizeof(int), hipMemcpyHostToDevice);
        
        // Call Kernel Function
        matrix_mult_gpu<<<grid_m, threads_m>>>(device_memory_a, device_memory_b, device_memory_c, M, N_1, K);

        // Copy back the results
        hipMemcpy(host_memory_c, device_memory_c, M * K * sizeof(int), hipMemcpyDeviceToHost);

        // Verify 
        transfer_to_matrix(host_memory_c, matrix_C);
        std::cout << "Matrix c (res) " << matrix_C->name << std::endl;
        printMatrix(matrix_C);

        free(host_memory_a);
        free(host_memory_b);
        free(host_memory_c);
        hipFree(device_memory_a);
        hipFree(device_memory_b);
        hipFree(device_memory_c);

        return matrix_C;
    }  
}
/* int rows, cols;
    std::tie(rows, cols) = m->dimension;
    for(int i = 0; i < rows; i++)
    {
        for(int j = 0; j < cols; j++)
        {
            std::cout << m->values[i][j] << " ";
        }
        std::cout << std::endl;
    }*/
void Sequence::transfer_to_ptr(int *a, matrix *x)
{
    int row, col;
    int counter = 0;
    std::tie(row, col) = x->dimension;
    for(int i = 0; i < row; i++)
    {
        for(int j = 0; j < col; j++)
        {
            a[counter] = x->values[i][j];
            counter+=1;
        }
    }
}

void Sequence::transfer_to_matrix(int *a, matrix *x)
{
    int row, col;
    std::tie(row, col) = x->dimension;
    for(int i = 0; i < row; i++)
    {
        for(int j = 0; j < col; j++)
        {
            x->values[i][j] = a[i*j];
        }
    } 
}