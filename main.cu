#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <tuple>
#include <vector>
#include <unordered_map>
#include <hip/hip_runtime.h>

#include <string.h>

struct matrix
{
    std::tuple<int, int> dimension;
};


struct pair_hash 
{
    std::size_t operator () (const std::pair<int,int> &p) const
    {
        return std::hash<int>{}(p.first) ^ std::hash<int>{}(p.second);
    }
};

/*
std::size_t unordered_map_hashing1(const std::pair<int, int>& p) {
    auto h1 = std::hash<int>{}(p.first);
    auto h2 = std::hash<int>{}(p.second);

    // A simple combination hash function
    return h1 ^ h2;
};
*/
int dp(int i, int j, std::vector<int> dimensions, std::vector<std::vector<int>> *s_table,  std::unordered_map<std::pair<int,int>,int, pair_hash> *seen)
{
    /*
        Base cases
    */
    if(i == j)
    {
        return 0;
    }
    std::pair<int,int> pair = std::make_pair(i,j);
    if(seen->find((pair)) != seen->end())
    {
        return (*seen)[pair];    
    }
    int res = 1000000;
    int s_value = 0;
    for(int k = i; k < j; k++)
    {
        int x = dp(i,k,dimensions, s_table, seen);
        int y = dp(k+1, j,dimensions, s_table, seen);
        int z = dimensions[i-1] * dimensions[k] * dimensions[j];
        int ans = x + y + z;
        if(ans < res)
        {
            res = ans;
            s_value = k;
        }
    }
    (*s_table)[i-1][j-1] = s_value;
    (*seen)[pair] = res;
    return res;

}

//Perform DP to find the path of least operations
void init_dp(std::vector<std::vector<int>> *s_table, std::vector<matrix> list_matrixes)
{
    /*
        Create dictionary for M_table called 'seen'

        -In C++ you need to provide a hashing function if you are 
        using a custom key 
    */
    std::unordered_map<std::pair<int,int>,int, pair_hash> seen;
    for(int i = 0; i < list_matrixes.size(); i++)
    {
        seen[std::make_pair(i,i)] = 0;
    }

    /*
        Get all dimenisons
    */
    std::vector<int> dimenions;
    for(int i = 0; i < list_matrixes.size(); i++)
    {
        matrix temp = list_matrixes[i];
        int a = std::get<0>(temp.dimension);
        int b = std::get<1>(temp.dimension);
        if(i == list_matrixes.size()-1)
        {
            dimenions.push_back(a);
            dimenions.push_back(b); 
        }
        else
        {
            dimenions.push_back(a);
        }
    }
    /*
    for(int i = 0; i < dimenions.size(); i++)
    {
        std::cout << dimenions[i] << " \n " << std::endl;
    }
    */
    dp(1,list_matrixes.size(), dimenions, s_table, &seen);
    std::cout << "max is " << seen[{1,4}] << std::endl;

    std::cout << "S TABLE: " << std::endl;
    for(int i = 0; i < s_table->size(); i++)
    {
        for(int j = 0; j < s_table[0].size(); j++)
        {
            std::cout << (*s_table)[i][j] << "\t";
        }
        std::cout << std::endl;
    }
}

int main()
{
    matrix A1;
    matrix A2;
    matrix A3;
    matrix A4;
    A1.dimension = std::make_tuple(5,4);
    A2.dimension = std::make_tuple(4,6);
    A3.dimension = std::make_tuple(6,2);
    A4.dimension = std::make_tuple(2,7);



    /*
        Create M and S tables
    */
    std::vector<std::vector<int>> s_table;

    int rows = 4;
    int columns = 4;
    for(int i = 0; i < rows; i++)
    {
        std::vector<int> row(columns, 0);
        s_table.push_back(row);
    }
    /*
        Initialize m_table to set self X self to be 0
    */

    /*
        Extract D_0 -> D_(N-1) values
    */
    std::vector<matrix> list_matrixes;
    list_matrixes.push_back(A1);
    list_matrixes.push_back(A2);
    list_matrixes.push_back(A3);
    list_matrixes.push_back(A4);
    init_dp(&s_table, list_matrixes);
    return 0;
}