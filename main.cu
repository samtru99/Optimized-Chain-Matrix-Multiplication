#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <tuple>
#include <vector>
#include <unordered_map>
#include <hip/hip_runtime.h>
#include <>
#include <string.h>
#include "sequence.h"
#include "helper_functions.h"



__global__ void GPUmatrixmult(int *a, int *b, int *c, int M, int N, int K)
{
   int row = blockIdx.y * blockDim.y + threadIdx.y;
   int col = blockIdx.x * blockDim.x + threadIdx.x;

   if(row < M && col < K)
   {
    int temp = 0;
    for(int i = 0; i < N; i++)
    {
        temp += a[row * N + i] * b[i*K+col];
    }
    //c[row*K+col] = temp;
    c[row*K+col] = 4;
   }
}

matrix* matrix_mult(matrix *a, matrix *b)
{
    matrix *c = (matrix*)malloc(sizeof(matrix));

    std::cout << "Matrix A is " << std::endl;
    print_matrix(a);
    std::cout << "Matrix B is "<< std::endl;
    print_matrix(b);

    int row_A,col_A,row_B,col_B;
    row_A = a->row;
    col_A = a->col;

    row_B = b->row; 
    col_B = b->col;
   
    c->row = row_A;
    c->col = col_B;
    c->ptr = (int*)malloc(row_A*col_B*sizeof(int));

    size_t bytes = row_A * col_B * sizeof(int);
    int *C;
    C = (int*)malloc(bytes);
    //C = (int*)malloc(row_A*col_B*sizeof(int));
    //Memory Allocations
    size_t bytes_A = row_A * col_A * sizeof(int);
    size_t bytes_B = row_B * col_B * sizeof(int);
    size_t bytes_C = row_A * col_B * sizeof(int);

    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes_A);
    hipMalloc(&d_b, bytes_B);
    hipMalloc(&d_c,bytes_C);

    hipMemcpy(d_a, a->ptr, bytes_A, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b->ptr, bytes_B, hipMemcpyHostToDevice);
    //hipMemcpy(d_c, &c->ptr, bytes_C, hipMemcpyHostToDevice);

    int threads = 4;
    int num_of_block_row = (int)ceil((row_A + threads - 1) / threads);
    int num_of_block_col = (int)ceil((col_B + threads - 1) / threads);

    dim3 THREADS(threads,threads);
    dim3 GRID(num_of_block_col,num_of_block_row);

    GPUmatrixmult<<<THREADS,GRID>>>(d_a,d_b,d_c,row_A,col_A,col_B);
    hipDeviceSynchronize();
    std::cout << "printing C MATRIX" << std::endl;
    hipMemcpy(C,d_c,bytes_C,hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    for(int i = 0; i < c->row; i++)
    {
        for(int j = 0; j < c->col;j++)
        {
            std::cout << C[i* c->col + j] << " ";
        }
        std::cout << std::endl;
    }
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return c;
}

matrix* compute(Node* n,std::unordered_map<char, matrix*>& dict )
{
    if(n->left == nullptr && n->right == nullptr && n->seq[1] == '\0') 
    { 
        return dict[n->seq[0]]; 
    }  
    if(n->left == nullptr && n->right == nullptr && n->seq[2] == '\0')  
    { 
        std::cout << "Computing Pair " << n->seq << std::endl; 
        matrix* matrix_A = dict[n->seq[0]];
        matrix* matrix_B = dict[n->seq[1]]; 
        std::cout << "Result is " << std::endl;
        matrix* matrix_C = matrix_mult(matrix_A,matrix_B); 
        //print_matrix(matrix_C);
        return matrix_C; 
    } 
    else 
    { 
        matrix* left_res = compute(n->left, dict); 
        matrix* right_res = compute(n->right, dict); 
        std::cout << "Node " << n->seq << " is " << std::endl;  
        matrix* matrix_C = matrix_mult(left_res,right_res);  
        return matrix_C;   
    }  
}  
 


int main()
{
    

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        std::cerr << "No CUDA-enabled devices found" << std::endl;
        return 1;
    }
    //Generate Matrices
    //Odd Number 
    matrix *a1 = new_matrix(4,10,1);
    matrix *a2 = new_matrix(10,3,2);
    matrix *a3 = new_matrix(3,12,3);
    matrix *a4 = new_matrix(12,20,4);
    matrix *a5 = new_matrix(20,7,5);

    std::unordered_map<char, matrix*> dict;
    dict['1'] = a1;
    dict['2'] = a2;
    dict['3'] = a3;
    dict['4'] = a4;
    dict['5'] = a5;

    std::vector<matrix> list_matrixes;
    list_matrixes.push_back(*a1);
    list_matrixes.push_back(*a2);
    list_matrixes.push_back(*a3);
    list_matrixes.push_back(*a4);
    list_matrixes.push_back(*a5);
    /*
        Create S table and initalize diagonal to zeros
    */
    std::vector<std::vector<int>> s_table;
    int rows = list_matrixes.size();
    int columns = list_matrixes.size();
    for(int i = 0; i < rows; i++)
    {
        std::vector<int> row(columns, 0);
        s_table.push_back(row);
    }
    /*
        Perform DP
    */
    init_dp(&s_table, list_matrixes);
    std::cout << "S TABLE: " << std::endl;
    for(int i = 0; i < s_table.size(); i++)
    {
        for(int j = 0; j < s_table[0].size(); j++)
        {
            std::cout << s_table[i][j] << "\t";
        }
        std::cout << std::endl;
    }
    
    Sequence seq(s_table, dict);
    Node* root = new Node();
    root = seq.init_sequence(1, list_matrixes.size());
     
    std::cout << "printing " << std::endl;
    std::cout << "(" << " ";
    seq.print_sequence(root);
    std::cout << ")" << " \n";
    
    std::cout << "COMPUTING " << std::endl;
    matrix* res = compute(root, dict);

    std::cout << "RES IS " << std::endl;
    seq.printMatrix(res);
    return 0;
}