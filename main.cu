#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <tuple>
#include <vector>
#include <unordered_map>
#include <hip/hip_runtime.h>
#include <>
#include <string.h>
#include "sequence.h"
#include "helper_functions.h"


__global__ void matrixMult()
{
 
}

int main()
{
    
    //Generate Matrices
    
    matrix *A1 = new matrix;
    matrix *A2 = new matrix;
    matrix *A3 = new matrix;
    matrix *A4 = new matrix;
    matrix *A5 = new matrix;


    matrix *a1 = new_matrix(4,10,1);
    matrix *a2 = new_matrix(10,3,2);
    matrix *a3 = new_matrix(3,12,3);
    matrix *a4 = new_matrix(12,20,4);
    matrix *a5 = new_matrix(20,7,5);
  
    //For odd number
    A1->dimension = std::make_tuple(4,10);
    A2->dimension = std::make_tuple(10,3);
    A3->dimension = std::make_tuple(3,12);
    A4->dimension = std::make_tuple(12,20);
    A5->dimension = std::make_tuple(20,7);
    


    std::cout << "1st matrix " << std::endl;
    setValues(A1, 1);
    //seq.printMatrix(A1);
    std::cout << "2nd matrix " << std::endl;
    setValues(A2, 2);
    //seq.printMatrix(A2);
    std::cout << "3rd matrix " << std::endl;
    setValues(A3, 3);
    std::cout << "4th matrix " << std::endl;
    setValues(A4, 4);
    std::cout << "5th matrix " << std::endl;
    setValues(A5, 5);
    //For even number 
    /*
    A1->dimension = std::make_tuple(3,2);
    A2->dimension = std::make_tuple(2,4);
    A3->dimension = std::make_tuple(4,2);
    A4->dimension = std::make_tuple(2,5);
    */
    std::unordered_map<char, matrix*> dict;
    dict['1'] = A1;
    dict['2'] = A2;
    dict['3'] = A3;
    dict['4'] = A4;
    dict['5'] = A5;

    std::vector<matrix> list_matrixes;
    list_matrixes.push_back(*A1);
    list_matrixes.push_back(*A2);
    list_matrixes.push_back(*A3);
    list_matrixes.push_back(*A4);
    list_matrixes.push_back(*A5);
    /*
        Create S table and initalize diagonal to zeros
    */
    std::vector<std::vector<int>> s_table;
    int rows = list_matrixes.size();
    int columns = list_matrixes.size();
    for(int i = 0; i < rows; i++)
    {
        std::vector<int> row(columns, 0);
        s_table.push_back(row);
    }
    /*
        Perform DP
    */
    init_dp(&s_table, list_matrixes);
    std::cout << "S TABLE: " << std::endl;
    for(int i = 0; i < s_table.size(); i++)
    {
        for(int j = 0; j < s_table[0].size(); j++)
        {
            std::cout << s_table[i][j] << "\t";
        }
        std::cout << std::endl;
    }
    
    Sequence seq(s_table, dict);

    

   

    Node* root = new Node();
    root = seq.init_sequence(1, list_matrixes.size());
     
    root = seq.init_sequence(1, list_matrixes.size());
    std::cout << "printing " << std::endl;
    std::cout << "(" << " ";
    seq.print_sequence(root);
    std::cout << ")" << " \n";
    
    std::cout << "COMPUTING " << std::endl;
    //matrix* res = seq.compute(root);

    //std::cout << "RES IS " << std::endl;
    //seq.printMatrix(res);
    return 0;
}