
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <tuple>
#include <vector>

struct matrix
{
    std::tuple<int, int> dimension;
};


int main()
{
    std::cout << "hello world " << std::endl;
    matrix A1;
    matrix A2;
    matrix A3;
    matrix A4;
    A1.dimension = std::make_tuple(5,4);
    A2.dimension = std::make_tuple(4,6);
    A3.dimension = std::make_tuple(6,2);
    A4.dimension = std::make_tuple(2,7);



    /*
        Create M and S tables
    */
    std::vector<std::vector<int>> m_table;
    std::vector<std::vector<int>> s_table;

    int rows = 4;
    int columns = 4;
    for(int i = 0; i < rows; i++)
    {
        std::vector<int> row(columns, 0);
        m_table.push_back(row);
        s_table.push_back(row);
    }
    /*
        Initialize m_table to set self X self to be 0
    */
    for(int i = 0; i < rows; i++)
    {
        m_table[i][i] = 0;
    }

    /*
        Extract D_0 -> D_(N-1) values
    */
    std::vector<matrix> list_matrixes;
    list_matrixes.push_back(A1);
    list_matrixes.push_back(A2);
    list_matrixes.push_back(A3);
    list_matrixes.push_back(A4);
    std::vector<int> dimenions;
    for(int i = 0; i < list_matrixes.size(); i++)
    {
        matrix temp = list_matrixes[i];
        int a = std::get<0>(temp.dimension);
        int b = std::get<1>(temp.dimension);
        if(i == list_matrixes.size()-1)
        {
            dimenions.push_back(a);
            dimenions.push_back(b);
        }
        else
        {
            dimenions.push_back(a);
        }
    }
    for(int i = 0; i < dimenions.size(); i++)
    {
        std::cout << dimenions[i] << " \n " << std::endl;
    }
    return 0;
}