#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <tuple>
#include <vector>
#include <unordered_map>
#include <hip/hip_runtime.h>
#include <>
#include <string.h>
#include "sequence.h"



struct pair_hash 
{
    std::size_t operator () (const std::pair<int,int> &p) const
    {
        return std::hash<int>{}(p.first) ^ std::hash<int>{}(p.second);
    }
};

int dp(int i, int j, std::vector<int> dimensions, std::vector<std::vector<int>> *s_table,  std::unordered_map<std::pair<int,int>,int, pair_hash> *seen)
{
    /*
        Base cases
    */
    if(i == j)
    {
        return 0;
    }
    std::pair<int,int> pair = std::make_pair(i,j);
    if(seen->find((pair)) != seen->end())
    {
        return (*seen)[pair];    
    }
    int res = 1000000;
    int s_value = 0;
    for(int k = i; k < j; k++)
    {
        int x = dp(i,k,dimensions, s_table, seen);
        int y = dp(k+1, j,dimensions, s_table, seen);
        int z = dimensions[i-1] * dimensions[k] * dimensions[j];
        int ans = x + y + z;
        if(ans < res)
        {
            res = ans;
            s_value = k;
        }
    }
    (*s_table)[i-1][j-1] = s_value;
    (*seen)[pair] = res;
    return res;

}

void setValues(matrix *x,int value)
{
    int row = std::get<0>(x->dimension);
    int col = std::get<1>(x->dimension);
    x->values.resize(row);
    for (int i = 0; i < row; ++i) 
    {
        x->values[i].resize(col);
    }
    for(int i = 0; i < row; i++)
    {
        for(int j = 0; j < col; j++)
        {
            x->values[i][j] = value;
        }
    }
}


//Perform DP to find the path of least operations
void init_dp(std::vector<std::vector<int>> *s_table, std::vector<matrix> list_matrixes)
{
    /*
        Create dictionary for M_table called 'seen'

        -In C++ you need to provide a hashing function if you are 
        using a custom key 
    */
    std::unordered_map<std::pair<int,int>,int, pair_hash> seen;
    for(int i = 0; i < list_matrixes.size(); i++)
    {
        seen[std::make_pair(i,i)] = 0;
    }

    /*
        Extract D_0 -> D_(N-1) dimenison values
    */
    std::vector<int> dimenions;
    for(int i = 0; i < list_matrixes.size(); i++)
    {
        matrix temp = list_matrixes[i];
        int a = std::get<0>(temp.dimension);
        int b = std::get<1>(temp.dimension);
        if(i == list_matrixes.size()-1)
        {
            dimenions.push_back(a);
            dimenions.push_back(b); 
        }
        else
        {
            dimenions.push_back(a);
        }
    }
    dp(1,list_matrixes.size(), dimenions, s_table, &seen);
    //std::cout << "MAX is " << seen[{1,list_matrixes.size()}] << std::endl;

}

int main()
{
    /*
        Generate Matrices
    */
    matrix *A1 = new matrix;
    matrix *A2 = new matrix;
    matrix *A3 = new matrix;
    matrix *A4 = new matrix;
    //matrix *A5 = new matrix;
    
    /*
    //For odd number 
    A1->dimension = std::make_tuple(4,10);
    A2->dimension = std::make_tuple(10,3);
    A3->dimension = std::make_tuple(3,12);
    A4->dimension = std::make_tuple(12,20);
    A5->dimension = std::make_tuple(20,7);
    */

    //For even number 
    A1->dimension = std::make_tuple(3,2);
    A2->dimension = std::make_tuple(2,4);
    A3->dimension = std::make_tuple(4,2);
    A4->dimension = std::make_tuple(2,5);
    setValues(A1, 1);
    setValues(A2, 2);
    setValues(A3, 3);
    setValues(A4, 4);
    //setValues(A5, 5);
    
   

    std::unordered_map<char, matrix*> dict;
    dict['1'] = A1;
    dict['2'] = A2;
    dict['3'] = A3;
    dict['4'] = A4;
    //dict['5'] = A5;

    std::vector<matrix> list_matrixes;
    list_matrixes.push_back(*A1);
    list_matrixes.push_back(*A2);
    list_matrixes.push_back(*A3);
    list_matrixes.push_back(*A4);
    //list_matrixes.push_back(*A5);


    /*
        Create S table and initalize diagonal to zeros
    */
    std::vector<std::vector<int>> s_table;
    int rows = list_matrixes.size();
    int columns = list_matrixes.size();
    for(int i = 0; i < rows; i++)
    {
        std::vector<int> row(columns, 0);
        s_table.push_back(row);
    }

    /*
        Perform DP
    */
    init_dp(&s_table, list_matrixes);

    //Print S-Table for debugging
    /*
    std::cout << "S TABLE: " << std::endl;
    for(int i = 0; i < s_table.size(); i++)
    {
        for(int j = 0; j < s_table[0].size(); j++)
        {
            std::cout << s_table[i][j] << "\t";
        }
        std::cout << std::endl;
    }
    */
    Sequence seq(s_table, dict);

    Node* root = new Node();
    root = seq.init_sequence(1, list_matrixes.size());
     
    root = seq.init_sequence(1, list_matrixes.size());
    std::cout << "printing " << std::endl;
    std::cout << "(" << " ";
    seq.print_sequence(root);
    std::cout << ")" << " \n";
    
    std::cout << "COMPUTING " << std::endl;
    matrix* res = seq.compute(root);

    std::cout << "RES IS " << std::endl;
    seq.printMatrix(res);
    return 0;
}